#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <math.h>
#include "../headers/graph.h"

#define THREADS_PER_BLOCK_X 32
#define THREADS_PER_BLOCK_Y 32

#define MAX_THREADS_PER_BLOCK 1024


using namespace std;


__global__
void AloopFW_inner(int ** d_x, int x_row_st, int x_col_st, 
			int u_row_st, int u_col_st, 
			int v_row_st, int v_col_st,
			int m, int k){

	

	int row_offset = blockIdx.x*blockDim.x + threadIdx.x;
	int col_offset = blockIdx.y*blockDim.y + threadIdx.y;

	int sum = d_x[u_row_st + row_offset][u_col_st + k] + d_x[v_row_st + k][v_col_st + col_offset];
	if(d_x[x_row_st + row_offset][x_col_st + col_offset] > sum)
		d_x[x_row_st + row_offset][x_col_st + col_offset] = sum;
	

	
	/*
	int rowsPerThread = m / blockDim.x;
	int colsPerThread = m / blockDim.y;

	int r_offset_start = threadIdx.x * rowsPerThread;
	int r_offset_end = r_offset_start + rowsPerThread - 1;

	int c_offset_start = threadIdx.y * colsPerThread;
	int c_offset_end = c_offset_start + colsPerThread - 1;

	for(int i = r_offset_start; i <= r_offset_end; i++){
		for(int j = c_offset_start; j <= c_offset_end; j++){
			int sum = d_x[u_row_st + i][u_col_st + k] + d_x[v_row_st + k][v_col_st + j];
			if(d_x[x_row_st + i][x_col_st + j] > sum)
				d_x[x_row_st + i][x_col_st + j] = sum;
		}
	}
	*/
}

//Called from host (outermost for loop)
void AloopFW_outer(int ** d_x, int x_row_st, int x_col_st, 
			int u_row_st, int u_col_st, 
			int v_row_st, int v_col_st,
			int m){

	int i,j,k; 
	for(k = 0; k < m; k++){	

		int threadX = min(m, THREADS_PER_BLOCK_X);
		int threadY = min(m, THREADS_PER_BLOCK_Y);

		int blocksX = m % threadX == 0 ? m/threadX : m/threadX + 1;
		int blocksY = m % threadY == 0 ? m/threadY : m/threadY + 1;

		dim3 blocksPerGrid(blocksX, blocksY);
		dim3 threadsPerBlock(threadX, threadY);
		AloopFW_inner<<<blocksPerGrid, threadsPerBlock>>>(d_x, x_row_st, x_col_st, u_row_st, u_col_st, v_row_st, v_col_st, m, k);
	}

}


__global__
void DloopFW(int ** d_x, int x_row_st, int x_col_st, 
			int u_row_st, int u_col_st, 
			int v_row_st, int v_col_st,
			int m){

	/*
	int kPerThread = m / blockDim.z;
	int rowsPerThread = m / blockDim.x;
	int colsPerThread = m / blockDim.y;

	int k_offset_start = threadIdx.z * kPerThread;
	int k_offset_end = k_offset_start + kPerThread - 1;

	int r_offset_start = threadIdx.x * rowsPerThread;
	int r_offset_end = r_offset_start + rowsPerThread - 1;

	int c_offset_start = threadIdx.y * colsPerThread;
	int c_offset_end = c_offset_start + colsPerThread - 1;

	for(int k = k_offset_start; k <= k_offset_end; k++){
		for(int i = r_offset_start; i <= r_offset_end; i++){
			for(int j = c_offset_start; j <= c_offset_end; j++){
				
				int sum = d_x[u_row_st + i][u_col_st + k] + d_x[v_row_st + k][v_col_st + j];
				if(d_x[x_row_st + i][x_col_st + j] > sum)
					d_x[x_row_st + i][x_col_st + j] = sum;
			}
		}
	}
	*/

	
	int k = blockIdx.z*blockDim.z + threadIdx.z;
	
	int row_offset = blockIdx.x*blockDim.x + threadIdx.x;
	int col_offset = blockIdx.y*blockDim.y + threadIdx.y;

	int sum = d_x[u_row_st + row_offset][u_col_st + k] + d_x[v_row_st + k][v_col_st + col_offset];
	if(d_x[x_row_st + row_offset][x_col_st + col_offset] > sum)
		d_x[x_row_st + row_offset][x_col_st + col_offset] = sum;

} 

//Recursive-3 implementation in HW1

void DFW(int ** x, int x_row_st, int x_col_st, 
			int u_row_st, int u_col_st, 
			int v_row_st, int v_col_st,
			int n, int m){

		if(m > n)
			return;

		if(n == m){		
			/*
			int threadZ = 1;
			int threadX = min(m, THREADS_PER_BLOCK_X);
			int threadY = min(m, THREADS_PER_BLOCK_Y);

			int blockZ = m;
			int blockX = m % threadX == 0 ? m/threadX : m/threadX + 1;
			int blockY = m % threadY == 0 ? m/threadY : m/threadY + 1;

			dim3 blocksPerGrid(blockX, blockY, blockZ);
			dim3 threadsPerBlock(threadX, threadY, threadZ);

			DloopFW<<<blocksPerGrid, threadsPerBlock>>>(x, x_row_st, x_col_st, u_row_st, u_col_st, v_row_st, v_col_st, m);
			*/

			AloopFW_outer(x, x_row_st, x_col_st, u_row_st, u_col_st, v_row_st, v_col_st, m);
		}
		else{
			int mid = n/2;
			//DFW (X11, U11, V11)
			DFW(x, x_row_st, x_col_st, u_row_st, u_col_st, v_row_st, v_col_st, mid, m);
			
			//DFW (X12, U11, V12)
			DFW(x, x_row_st, x_col_st + mid, u_row_st, u_col_st, v_row_st, v_col_st + mid, mid, m);
			
			//DFW (X21, U21, V11)
			DFW(x, x_row_st + mid, x_col_st, u_row_st + mid, u_col_st, v_row_st, v_col_st, mid, m);

			//DFW (X22, U21, V12)
			DFW(x, x_row_st + mid, x_col_st + mid, u_row_st + mid, u_col_st, v_row_st, v_col_st + mid, mid, m);

			//DFW (X11, U12, V21)
			DFW(x, x_row_st, x_col_st, u_row_st, u_col_st + mid, v_row_st + mid, v_col_st, mid, m);

			//DFW (X12, U12, V22)
			DFW(x, x_row_st, x_col_st + mid, u_row_st, u_col_st + mid, v_row_st + mid, v_col_st + mid, mid, m);

			//DFW (X21, U22, V21)
			DFW(x, x_row_st + mid, x_col_st, u_row_st + mid, u_col_st + mid, v_row_st + mid, v_col_st, mid, m);

			//DFW (X22, U22, V22)
			DFW(x, x_row_st + mid, x_col_st + mid, u_row_st + mid, u_col_st + mid, v_row_st + mid, v_col_st + mid, mid, m);
		}

		
}


void DFW(int ** d_x, int x_row_st, int x_col_st, 
			int u_row_st, int u_col_st, 
			int v_row_st, int v_col_st,
			int n, int depth, int * tilesize){
	int r = tilesize[depth];
	if(r > n){
		//Execute base case
		AloopFW_outer(d_x, x_row_st, x_col_st, u_row_st, u_col_st, v_row_st, v_col_st, n);
	}
	else
		printf("Here\n");

}


void CFW(int ** d_x, int x_row_st, int x_col_st, 
			int u_row_st, int u_col_st, 
			int v_row_st, int v_col_st,
			int n, int depth, int * tilesize){
	int r = tilesize[depth];
	if(r > n){
		//Execute base case
		AloopFW_outer(d_x, x_row_st, x_col_st, u_row_st, u_col_st, v_row_st, v_col_st, n);
	}
	else
		printf("Here\n");

}

void BFW(int ** d_x, int x_row_st, int x_col_st, 
			int u_row_st, int u_col_st, 
			int v_row_st, int v_col_st,
			int n, int depth, int * tilesize){
	int r = tilesize[depth];
	if(r > n){
		//Execute base case
		AloopFW_outer(d_x, x_row_st, x_col_st, u_row_st, u_col_st, v_row_st, v_col_st, n);
	}
	else
		printf("Here\n");
}



//Figure 4 implementation : HW 5
void AFW(int ** d_x, int x_row_st, int x_col_st, 
			int u_row_st, int u_col_st, 
			int v_row_st, int v_col_st,
			int n, int depth, int * tilesize){
	int r = tilesize[depth];
	if(r > n){
		//Execute base case
		AloopFW_outer(d_x, x_row_st, x_col_st, u_row_st, u_col_st, v_row_st, v_col_st, n);
	}
	else{
		int sub_size = n / r;
		for(int k = 0; k < r; k++){
			int offset = k*sub_size;
			AFW(d_x, x_row_st + offset, x_col_st + offset, u_row_st + offset, u_col_st + offset, v_row_st + offset, v_col_st + offset, sub_size, depth+1, tilesize);
			
			//SYNC POINT
			hipDeviceSynchronize();
			for(int j = 0; j < r; j++){
				if(j == k)
					continue;
				BFW(d_x, x_row_st + offset, x_col_st + j*sub_size, u_row_st + offset, u_col_st + offset, v_row_st + offset, v_col_st + j*sub_size, sub_size, depth+1, tilesize);
				CFW(d_x, x_row_st + j*sub_size, x_col_st + offset, u_row_st + j*sub_size, u_col_st + offset, v_row_st + offset, v_col_st + offset, sub_size, depth+1, tilesize);
			}
			hipDeviceSynchronize();

			for(int i = 0; i < r; i++){
				if(i == k)
					continue;
				for(int j = 0; j < r; j++){
					if(j == k)
						continue;
					DFW(d_x, x_row_st + i*sub_size, x_col_st + j*sub_size, u_row_st + i*sub_size, u_col_st + offset, v_row_st + offset, v_col_st + j*sub_size, sub_size, depth+1, tilesize);
				}
			}
		}
	}
}//AFW 






int ** copy_matrix_to_host(int ** dev_matrix, int n){
	int ** new_matrix = new int*[n+1];
	for(int i=1;i <= n; i++){
		new_matrix[i] = new int[n+1];
		int * begin;
		hipMemcpy(&begin, &dev_matrix[i], sizeof (int *), hipMemcpyDeviceToHost);
		hipMemcpy(new_matrix[i], begin, (n+1) * sizeof(int), hipMemcpyDeviceToHost);
	}
	return new_matrix;
}

int ** copy_matrix_to_device(int ** host_matrix, int n){
	//int ** dev_matrix = new int*[n+1];
	int ** dev_matrix;
	hipError_t err = hipMalloc(&dev_matrix, (n+1) * sizeof(int *));
	if(err != hipSuccess){
		printf("Error allocating memory on device.");
		return NULL;
	}
	for(int i = 1; i <= n; i++){
		//printf("%x\n", &addr[i]);
		int * start;
		err = hipMalloc(&start, (n+1)*sizeof(int));
		if(err != hipSuccess){
			printf("Error allocating memory on device.");
			return NULL;
		}
		hipMemcpy(dev_matrix+i, &start, sizeof(int *), hipMemcpyHostToDevice);
		hipMemcpy(start, host_matrix[i], (n+1) * sizeof(int), hipMemcpyHostToDevice);
	}
	return dev_matrix;
}

int main(int argc, char * argv[])
{
	
	//Matrix
	int n = atoi(argv[1]);
	int m = 4;
	int ** matrix = generate_matrix(n);
	int ** dev_matrix = copy_matrix_to_device(matrix, n);
	
	if(dev_matrix == NULL)
		return 0;	

	// fw_iterative_outer(dev_matrix, n);
	printf("Original matrix: \n");
	print_matrix(matrix, n);
	
	long long start, end;
	start = clock();
	int tilesize[2] = {4, INT_MAX};
	AFW(dev_matrix, 1, 1, 1, 1, 1, 1, n, 0, tilesize);
	end = clock();
	int ** new_matrix = copy_matrix_to_host(dev_matrix, n);
	printf("\nWith updated distances: \n");
	print_matrix(new_matrix, n);
	//cout << "Runtime: " << double(end-start)/double(CLOCKS_PER_SEC) << endl;
	return 0;
}






