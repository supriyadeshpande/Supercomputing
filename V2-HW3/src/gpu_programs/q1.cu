#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <math.h>
#include "../headers/graph.h"

#define THREADS_PER_BLOCK_X 32
#define THREADS_PER_BLOCK_Y 32

#define MAX_THREADS_PER_BLOCK 1024


using namespace std;


__global__
void AloopFW(int ** d_x, int x_row_st, int x_col_st, 
			int u_row_st, int u_col_st, 
			int v_row_st, int v_col_st,
			int m){

	int tx = threadIdx.x, ty = threadIdx.y, sum;

	int rowsPerThread = m / blockDim.x;
	int colsPerThread = m / blockDim.y;

	int r_offset_start = threadIdx.x * rowsPerThread;
	int r_offset_end = r_offset_start + rowsPerThread - 1;

	int c_offset_start = threadIdx.y * colsPerThread;
	int c_offset_end = c_offset_start + colsPerThread - 1;

	for(int k = 0; k < m; k++){

		if(tx == 0 && ty == 0){
			//update cell (k,k)
			sum = d_x[u_row_st + k][u_col_st + k] + d_x[v_row_st + k][v_col_st + k];
			d_x[x_row_st + k][x_col_st + k] = d_x[x_row_st + k][x_col_st + k] > sum ? sum : d_x[x_row_st + k][x_col_st + k];	
		}
			
		syncthreads();

		//Thread X responsible for updating current row.
		if(r_offset_start <= k && k<= r_offset_end){
			for(int j = c_offset_start; j <= c_offset_end; j++){
				if(j == k)
					continue;
				sum = d_x[u_row_st + k][u_col_st + k] + d_x[v_row_st + k][v_col_st + j];
				if(d_x[x_row_st + k][x_col_st + j] > sum)
					d_x[x_row_st + k][x_col_st + j] = sum;
			}
		}

		syncthreads();

		//Thread Y responsible for updating current column
		if(c_offset_start <= k && k <= c_offset_end){
			for(int i = r_offset_start; i <= r_offset_end; i++){
				if(i == k)
					continue;
				sum = d_x[u_row_st + i][u_col_st + k] + d_x[v_row_st + k][v_col_st + k];
				if(d_x[x_row_st + i][x_col_st + k] > sum)
					d_x[x_row_st + i][x_col_st + k] = sum;
			}
		}

		syncthreads();

		for(int i = r_offset_start; i <= r_offset_end; i++){
			if(i == k)
				continue;
			for(int j = c_offset_start; j <= c_offset_end; j++){
				if(j == k)
					continue;
				int sum = d_x[u_row_st + i][u_col_st + k] + d_x[v_row_st + k][v_col_st + j];
				if(d_x[x_row_st + i][x_col_st + j] > sum)
					d_x[x_row_st + i][x_col_st + j] = sum;
			}
		}
		syncthreads();

	}
	
}

__global__
void BloopFW(int ** d_x, int x_row_st, int x_col_st, 
			int u_row_st, int u_col_st, 
			int v_row_st, int v_col_st,
			int m){

	//int tx = threadIdx.x, ty = threadIdx.y;
	int sum;

	int rowsPerThread = m / blockDim.x;
	int colsPerThread = m / blockDim.y;

	int r_offset_start = threadIdx.x * rowsPerThread;
	int r_offset_end = r_offset_start + rowsPerThread - 1;

	int c_offset_start = threadIdx.y * colsPerThread;
	int c_offset_end = c_offset_start + colsPerThread - 1;	

	for(int k=0; k < m; k++){

		//Update kth row using the corresponding thread.
		if(r_offset_start <= k && k<= r_offset_end){
			for(int j = c_offset_start; j <= c_offset_end; j++){				
				sum = d_x[u_row_st + k][u_col_st + k] + d_x[v_row_st + k][v_col_st + j];
				if(d_x[x_row_st + k][x_col_st + j] > sum)
					d_x[x_row_st + k][x_col_st + j] = sum;
			}
		}

		syncthreads();

		//Update the other cells.
		for(int i = r_offset_start; i <= r_offset_end; i++){
			if(i == k)
				continue;
			for(int j = c_offset_start; j <= c_offset_end; j++){
				int sum = d_x[u_row_st + i][u_col_st + k] + d_x[v_row_st + k][v_col_st + j];
				if(d_x[x_row_st + i][x_col_st + j] > sum)
					d_x[x_row_st + i][x_col_st + j] = sum;
			}
		}

		syncthreads();
	}
}


__global__
void CloopFW(int ** d_x, int x_row_st, int x_col_st, 
			int u_row_st, int u_col_st, 
			int v_row_st, int v_col_st,
			int m){

	//int tx = threadIdx.x, ty = threadIdx.y;
	int sum;

	int rowsPerThread = m / blockDim.x;
	int colsPerThread = m / blockDim.y;

	int r_offset_start = threadIdx.x * rowsPerThread;
	int r_offset_end = r_offset_start + rowsPerThread - 1;

	int c_offset_start = threadIdx.y * colsPerThread;
	int c_offset_end = c_offset_start + colsPerThread - 1;	

	for(int k=0; k < m; k++){

		if(c_offset_start <= k && k <= c_offset_end){
			for(int i = r_offset_start; i <= r_offset_end; i++){
				sum = d_x[u_row_st + i][u_col_st + k] + d_x[v_row_st + k][v_col_st + k];
				if(d_x[x_row_st + i][x_col_st + k] > sum)
					d_x[x_row_st + i][x_col_st + k] = sum;
			}
		}

		syncthreads();

		for(int i = r_offset_start; i <= r_offset_end; i++){
			for(int j = c_offset_start; j <= c_offset_end; j++){
				if(j == k)
					continue;
				int sum = d_x[u_row_st + i][u_col_st + k] + d_x[v_row_st + k][v_col_st + j];
				if(d_x[x_row_st + i][x_col_st + j] > sum)
					d_x[x_row_st + i][x_col_st + j] = sum;
			}
		}

		syncthreads();
	}	

}




__global__
void DloopFW(int ** d_x, int x_row_st, int x_col_st, 
			int u_row_st, int u_col_st, 
			int v_row_st, int v_col_st,
			int m){

	int rowsPerThread = m / blockDim.x;
	int colsPerThread = m / blockDim.y;

	int r_offset_start = threadIdx.x * rowsPerThread;
	int r_offset_end = r_offset_start + rowsPerThread - 1;

	int c_offset_start = threadIdx.y * colsPerThread;
	int c_offset_end = c_offset_start + colsPerThread - 1;
	
	for(int k = 0; k < m; k++){

		for(int i = r_offset_start; i <= r_offset_end; i++){
			for(int j = c_offset_start; j <= c_offset_end; j++){
				
				int sum = d_x[u_row_st + i][u_col_st + k] + d_x[v_row_st + k][v_col_st + j];
				if(d_x[x_row_st + i][x_col_st + j] > sum)
					d_x[x_row_st + i][x_col_st + j] = sum;
			}
		}
		syncthreads();
	}

} 

//Recursive-3 implementation in HW1

void DFW(int ** x, int x_row_st, int x_col_st, 
			int u_row_st, int u_col_st, 
			int v_row_st, int v_col_st,
			int n, int m){

		if(m > n)
			return;

		if(n == m){			
			
			int threadX = min(m, THREADS_PER_BLOCK_X);
			int threadY = min(m, THREADS_PER_BLOCK_Y);
			dim3 threadsPerBlock(threadX, threadY);

			DloopFW<<<1, threadsPerBlock>>>(x, x_row_st, x_col_st, u_row_st, u_col_st, v_row_st, v_col_st, m);
		}
		else{
			int mid = n/2;
			//DFW (X11, U11, V11)
			DFW(x, x_row_st, x_col_st, u_row_st, u_col_st, v_row_st, v_col_st, mid, m);
			
			//DFW (X12, U11, V12)
			DFW(x, x_row_st, x_col_st + mid, u_row_st, u_col_st, v_row_st, v_col_st + mid, mid, m);
			
			//DFW (X21, U21, V11)
			DFW(x, x_row_st + mid, x_col_st, u_row_st + mid, u_col_st, v_row_st, v_col_st, mid, m);

			//DFW (X22, U21, V12)
			DFW(x, x_row_st + mid, x_col_st + mid, u_row_st + mid, u_col_st, v_row_st, v_col_st + mid, mid, m);

			//DFW (X11, U12, V21)
			DFW(x, x_row_st, x_col_st, u_row_st, u_col_st + mid, v_row_st + mid, v_col_st, mid, m);

			//DFW (X12, U12, V22)
			DFW(x, x_row_st, x_col_st + mid, u_row_st, u_col_st + mid, v_row_st + mid, v_col_st + mid, mid, m);

			//DFW (X21, U22, V21)
			DFW(x, x_row_st + mid, x_col_st, u_row_st + mid, u_col_st + mid, v_row_st + mid, v_col_st, mid, m);

			//DFW (X22, U22, V22)
			DFW(x, x_row_st + mid, x_col_st + mid, u_row_st + mid, u_col_st + mid, v_row_st + mid, v_col_st + mid, mid, m);
		}

		
}

void CFW(int ** x, int x_row_st, int x_col_st, 
			int u_row_st, int u_col_st, 
			int v_row_st, int v_col_st,
			int n, int m){

	if(m > n)
		return;

	if(n == m){		
		int threadX = min(m, THREADS_PER_BLOCK_X);
		int threadY = min(m, THREADS_PER_BLOCK_Y);
		dim3 threadsPerBlock(threadX, threadY);

		CloopFW<<<1, threadsPerBlock>>>(x, x_row_st, x_col_st, u_row_st, u_col_st, v_row_st, v_col_st, m);
	}
	else{
		int mid = n/2;
		CFW(x, x_row_st, x_col_st, u_row_st, u_col_st, v_row_st, v_col_st, mid, m);		
		CFW(x, x_row_st + mid, x_col_st, u_row_st + mid, u_col_st, v_row_st, v_col_st, mid, m);

		DFW(x, x_row_st, x_col_st + mid, u_row_st, u_col_st, v_row_st, v_col_st + mid, mid, m);
		DFW(x, x_row_st + mid, x_col_st + mid, u_row_st + mid, u_col_st, v_row_st, v_col_st + mid, mid, m);

		CFW(x, x_row_st, x_col_st + mid, u_row_st, u_col_st + mid, v_row_st + mid, v_col_st + mid, mid, m);
		CFW(x, x_row_st + mid, x_col_st + mid, u_row_st + mid, u_col_st + mid, v_row_st + mid, v_col_st + mid, mid, m);

		DFW(x, x_row_st, x_col_st, u_row_st, u_col_st + mid, v_row_st + mid, v_col_st, mid, m);	
		DFW(x, x_row_st + mid, x_col_st, u_row_st + mid, u_col_st + mid, v_row_st + mid, v_col_st, mid, m);
	}
}

void BFW(int ** x, int x_row_st, int x_col_st, 
			int u_row_st, int u_col_st, 
			int v_row_st, int v_col_st,
			int n, int m){

	if(m > n)
		return;

	if(n == m){
		int threadX = min(m, THREADS_PER_BLOCK_X);
		int threadY = min(m, THREADS_PER_BLOCK_Y);
		dim3 threadsPerBlock(threadX, threadY);

		BloopFW<<<1, threadsPerBlock>>>(x, x_row_st, x_col_st, u_row_st, u_col_st, v_row_st, v_col_st, m);
	}
	else{
		int mid = n/2;
		BFW(x, x_row_st, x_col_st, u_row_st, u_col_st, v_row_st, v_col_st, mid, m);
		BFW(x, x_row_st, x_col_st + mid, u_row_st, u_col_st, v_row_st, v_col_st + mid, mid, m);

		DFW(x, x_row_st + mid, x_col_st, u_row_st + mid, u_col_st, v_row_st, v_col_st, mid, m);
		DFW(x, x_row_st + mid, x_col_st + mid, u_row_st + mid, u_col_st, v_row_st, v_col_st + mid, mid, m);

		BFW(x, x_row_st + mid, x_col_st, u_row_st + mid, u_col_st + mid, v_row_st + mid, v_col_st, mid, m);
		BFW(x, x_row_st + mid, x_col_st + mid, u_row_st + mid, u_col_st + mid, v_row_st + mid, v_col_st + mid, mid, m);

		DFW(x, x_row_st, x_col_st, u_row_st, u_col_st + mid, v_row_st + mid, v_col_st, mid, m);
		DFW(x, x_row_st, x_col_st + mid, u_row_st, u_col_st + mid, v_row_st + mid, v_col_st + mid, mid, m);
	}
}

//Recursive implementation (PARALLEL)
void AFW(int ** x, int x_row_st, int x_col_st, 
			int u_row_st, int u_col_st, 
			int v_row_st, int v_col_st,
			int n, int m){

	//Incase of wrong values entered at runtime.
	if(m > n)
		return;

	//Recursion base case
	if(n == m){
		int threadX = min(m, THREADS_PER_BLOCK_X);
		int threadY = min(m, THREADS_PER_BLOCK_Y);
		dim3 threadsPerBlock(threadX, threadY);

		AloopFW<<<1, threadsPerBlock>>>(x, x_row_st, x_col_st, u_row_st, u_col_st, v_row_st, v_col_st, m);	
	}
	else{
		int mid = n/2;

		//AFW (X11, U11, V11)
		AFW(x, x_row_st, x_col_st, u_row_st, u_col_st, v_row_st, v_col_st, mid, m);
		
		//BFW (X12, U11, V12)
		BFW(x, x_row_st, x_col_st + mid, u_row_st, u_col_st, v_row_st, v_col_st + mid, mid, m);

		//CFW (X21, U21, V11)
		CFW(x, x_row_st + mid, x_col_st, u_row_st + mid, u_col_st, v_row_st, v_col_st, mid, m);

		//DFW (X22, U21, V12)
		DFW(x, x_row_st + mid, x_col_st + mid, u_row_st + mid, u_col_st, v_row_st, v_col_st + mid, mid, m);

		//AFW (X22, U22, V22)
		AFW(x, x_row_st + mid, x_col_st + mid, u_row_st + mid, u_col_st + mid, v_row_st + mid, v_col_st + mid, mid, m);

		//BFW (X21, U22, V21)
		BFW(x, x_row_st + mid, x_col_st, u_row_st + mid, u_col_st + mid, v_row_st + mid, v_col_st, mid, m);

		//CFW (X12, U12, V22)
		CFW(x, x_row_st, x_col_st + mid, u_row_st, u_col_st + mid, v_row_st + mid, v_col_st + mid, mid, m);

		//DFW (X11, U12, V21)
		DFW(x, x_row_st, x_col_st, u_row_st, u_col_st + mid, v_row_st + mid, v_col_st, mid, m);
	}

}




int ** copy_matrix_to_host(int ** dev_matrix, int n){
	int ** new_matrix = new int*[n+1];
	for(int i=1;i <= n; i++){
		new_matrix[i] = new int[n+1];
		int * begin;
		hipMemcpy(&begin, &dev_matrix[i], sizeof (int *), hipMemcpyDeviceToHost);
		hipMemcpy(new_matrix[i], begin, (n+1) * sizeof(int), hipMemcpyDeviceToHost);
	}
	return new_matrix;
}

int ** copy_matrix_to_device(int ** host_matrix, int n){
	//int ** dev_matrix = new int*[n+1];
	int ** dev_matrix;
	hipMalloc(&dev_matrix, (n+1) * sizeof(int *));
	for(int i = 1; i <= n; i++){
		//printf("%x\n", &addr[i]);
		int * start;
		hipMalloc(&start, (n+1)*sizeof(int));
		hipMemcpy(dev_matrix+i, &start, sizeof(int *), hipMemcpyHostToDevice);
		hipMemcpy(start, host_matrix[i], (n+1) * sizeof(int), hipMemcpyHostToDevice);
	}
	return dev_matrix;
}

int main(int argc, char * argv[])
{
	
	//Matrix
	int n = atoi(argv[1]);
	int m = 4;
	int ** matrix = generate_matrix(n);
	int ** dev_matrix = copy_matrix_to_device(matrix, n);
	
	if(n <= 32){
		printf("Original matrix: \n");
		print_matrix(matrix, n);
	}

	long long start, end;
	start = clock();
	AFW(dev_matrix, 1, 1, 1, 1, 1, 1, n, m);
	end = clock();
	int ** new_matrix = copy_matrix_to_host(dev_matrix, n);
	if(n <= 32){
		printf("\nWith updated distances: \n");
		print_matrix(new_matrix, n);	
	}
	
	cout << "Runtime: " << double(end-start)/double(CLOCKS_PER_SEC) << endl;
	return 0;
}






